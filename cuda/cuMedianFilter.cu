#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : cuMedianFilter.cu
 Author      : Oded Green
 Version     :
 Copyright   : BSD 3-Clause
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>


#define TEST
// THREADS is defined to be the max threads
#define MAX_THREADS 256
#define SINGLE_BLOCK 0
#define PRINT_ON 0

int print_u, print_v;

typedef unsigned short hist_type;
typedef unsigned char im_type;

#define MF_IM_SIZE 512
#define MF_HIST_SIZE 256

#include "lookup.h"


#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)

#define max(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })
#define min(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _b : _a; })



__device__ void histogramAdd(hist_type* H, const hist_type * hist_col){
	int32_t tx = threadIdx.x;
	for(; tx<256;tx+=blockDim.x){
		H[tx]+=hist_col[tx];
	}
}

__device__ void histogramSub(hist_type* H, const hist_type * hist_col){
	int32_t tx = threadIdx.x;
	for(; tx<256;tx+=blockDim.x){
		H[tx]-=hist_col[tx];
	}
}

__device__ void histogramAddAndSub(hist_type* H, const hist_type * hist_colAdd,const hist_type * hist_colSub){
	int32_t tx = threadIdx.x;
	for(; tx<256;tx+=blockDim.x){
		H[tx]+=hist_colAdd[tx]-hist_colSub[tx];
	}
}

__device__ void histogramMultipleAdd(hist_type* H, const hist_type * hist_col,int histCount){
	int32_t tx = threadIdx.x;
	for(; tx<256;tx+=blockDim.x){
		hist_type temp=H[tx];;
		for(int i=0; i<histCount; i++)
		    temp+=hist_col[(i<<8)+tx];
		H[tx]=temp;
	}
}


__device__ void histogramClear(hist_type* H){
	int32_t tx = threadIdx.x;
	for(; tx<256;tx+=blockDim.x){
		H[tx]=0;
	}
}

__device__ void histogramClearAllColmuns(hist_type* hist, const int32_t columns,const int32_t hist_size){
	int32_t tx = threadIdx.x;
	int array_size=columns*hist_size;
	for(; tx<array_size;tx+=blockDim.x){
		hist[tx]=0;
	}
}




__device__ void histogramMedianPar64(hist_type* H,hist_type* Hscan,const int32_t size, const int32_t medPos, const int logSize,im_type* retval){
	int32_t tx=threadIdx.x;
	*retval=1;

	Hscan[tx]=H[tx];
	Hscan[tx+64]=H[tx+64];
	Hscan[tx+128]=H[tx+128];
	Hscan[tx+192]=H[tx+192];	
	syncthreads();

	int32_t add;

//	if(tx<128){
	add=tx<<1; Hscan[add+1]+=Hscan[add];
	add=64+tx<<1; Hscan[add+1]+=Hscan[add];

	syncthreads();
	if(tx<64){
		add=tx<<2; Hscan[add+3]+=Hscan[add+1];
	}
	syncthreads();

	if(tx<32){
		add=tx<<3;Hscan[add+7]+=Hscan[add+3];
	}
	if(tx<16){
		add=tx<<4; Hscan[add+15]+=Hscan[add+7];
	}
	if(tx<8){
		add=tx<<5; Hscan[add+31]+=Hscan[add+15];
	}
	if(tx<4){
		add=tx<<6; Hscan[add+63]+=Hscan[add+31];
	}
	if(tx<2){
		add=tx<<7; Hscan[add+127]+=Hscan[add+63];
	}
	Hscan[MF_HIST_SIZE-1]=0;
	syncthreads();

	hist_type temp;
	if (tx<1){
		add=tx<<8;
		temp=Hscan[add+127]; Hscan[add+127]=Hscan[add+255]; Hscan[add+255]+=temp;
	}
	if (tx<2){
		add=tx<<7;
		temp=Hscan[add+63]; Hscan[add+63]=Hscan[add+127]; Hscan[add+127]+=temp;
	}
	if (tx<4){
		add=tx<<6;
		temp=Hscan[add+31]; Hscan[add+31]=Hscan[add+63]; Hscan[add+63]+=temp;
	}
	if (tx<8){
		add=tx<<5;
		temp=Hscan[add+15]; Hscan[add+15]=Hscan[add+31]; Hscan[add+31]+=temp;
	}
	if (tx<16){
		add=tx<<4;
		temp=Hscan[add+7]; Hscan[add+7]=Hscan[add+15]; Hscan[add+15]+=temp;
	}
	if (tx<32){
		add=tx<<3;
		temp=Hscan[add+3]; Hscan[add+3]=Hscan[add+7]; Hscan[add+7]+=temp;
	}
	syncthreads();
	if (tx<64){
		add=tx<<2;
		temp=Hscan[add+1]; Hscan[add+1]=Hscan[add+3]; Hscan[add+3]+=temp;
	}
	syncthreads();
//	if (tx<128){
	{
		add=tx<<1;
		temp=Hscan[add+0]; Hscan[add+0]=Hscan[add+1]; Hscan[add+1]+=temp;
		
		add=64+tx<<1;
		temp=Hscan[add+0]; Hscan[add+0]=Hscan[add+1]; Hscan[add+1]+=temp;
		
	}
	syncthreads();
	tx=threadIdx.x;
	for(; tx<255;tx+=blockDim.x){
		if(Hscan[tx+1]>=medPos){
			if(Hscan[tx]<medPos){ 
				*retval=tx;
			}
			break;
			
		}
	}
	syncthreads();
}


__device__ void histogramMedianPar32Multi(hist_type* H,hist_type* Hscan,const int32_t size, const int32_t medPos, const int logSize,im_type* retval){
	int32_t tx=threadIdx.x;
	*retval=0;
	__shared__ int32_t foundIn;
	foundIn=31;
//	if(tx<32)
//	{
		Hscan[tx]=0;
		int add=tx<<3;
		for(int i=0; i<8;i++){
			Hscan[tx]+=H[add+i];
		}

		if(tx<16){
			add=tx<<1; Hscan[add+1]+=Hscan[add];
		}
		if(tx<8){
			add=tx<<2; Hscan[add+3]+=Hscan[add+1];
		}
		if(tx<4){
			add=tx<<3; Hscan[add+7]+=Hscan[add+3];
		}
		if(tx<2){
			add=tx<<4; Hscan[add+15]+=Hscan[add+7];
		}

		Hscan[31]=0;
		
		hist_type temp;
		if (tx<1){
			add=0;
			temp=Hscan[add+15]; Hscan[add+15]=Hscan[add+31]; Hscan[add+31]+=temp;
		}
		if (tx<2){
			add=tx<<4;
			temp=Hscan[add+7]; Hscan[add+7]=Hscan[add+15]; Hscan[add+15]+=temp;
		}
		if (tx<4){
			add=tx<<3;
			temp=Hscan[add+3]; Hscan[add+3]=Hscan[add+7]; Hscan[add+7]+=temp;
		}
		if (tx<8){
			add=tx<<2;
			temp=Hscan[add+1]; Hscan[add+1]=Hscan[add+3]; Hscan[add+3]+=temp;
		}
		if (tx<16){
			add=tx<<1;
			temp=Hscan[add]; Hscan[add]=Hscan[add+1]; Hscan[add+1]+=temp;
		}
	
		syncthreads();
		if(tx<31){
			if(Hscan[tx+1]>=medPos){
				if(Hscan[tx]<medPos){ 
					foundIn=tx;
				}
			}
		}	
//			if(blockIdx.x==0 && threadIdx.x==0){
//				printf("%d  %d  %d %d \n",Hscan[foundIn], Hscan[foundIn+1], foundIn,medPos);
//			}

		
		syncthreads();
		if(tx==0){
			int32_t pos=foundIn<<3;
			int32_t total=Hscan[foundIn]+H[pos];
			for(int i=0; i<8;i++)
			{
				if(total>=medPos){
					*retval=pos+i;
					break;
				}
				total+=H[pos+i+1];
			}
//			if(blockIdx.x==0){
//				for(int i=0; i<32; i++)
//					printf("%d ", Hscan[i]);
//				printf("      -  %d  %d  %d  %d\n",foundIn, foundIn<<3,Hscan[foundIn], total);
//			}
			
		}
//	}



	/*
	syncthreads();
	tx=threadIdx.x;
	for(; tx<255;tx+=blockDim.x){
		if(Hscan[tx+1]>=medPos){
			if(Hscan[tx]<medPos){ 
				*retval=tx;
			}
			break;
			
		}
	}
	syncthreads();
*/	
}

__device__ void histogramMedianPar32WorkInefficient(hist_type* H,hist_type* Hscan,const int32_t size, const int32_t medPos, const int logSize,im_type* retval){
	int32_t tx=threadIdx.x;
	*retval=0;
	__shared__ int32_t foundIn;
	foundIn=31;
        if (tx>32)
	  return;
	
	//if(tx<32)
	{
		  Hscan[tx]=0;
		int add=tx<<3;
		for(int i=0; i<8;i++){
	        	Hscan[tx]+=H[add+i];
		}
		syncthreads();
		if(tx>1){
		  Hscan[tx]+=Hscan[tx-1];
		}
 		if(tx>=2){
		  Hscan[tx]+=Hscan[tx-2];
		}
    		if(tx>=4){
		  Hscan[tx]+=Hscan[tx-4];
		}
 		if(tx>=8){
		  Hscan[tx]+=Hscan[tx-8];
		}
 		if(tx>=16){
		  Hscan[tx]+=Hscan[tx-16];
		}
 
	}
/*	       		if(blockIdx.x==0 && threadIdx.x==0){
				for(int i=0; i<32; i++)
					printf("%d ", Hscan[i]);
		//		printf("      -  %d \n",Hscan[i]);
		        	printf("\n");
			}
*/       
		syncthreads();
		if(tx<31){
			if(Hscan[tx+1]>=medPos){
				if(Hscan[tx]<medPos){ 
					foundIn=tx;
				}
			}
		}	
		syncthreads();
/*       	                         			if(blockIdx.x==0 && threadIdx.x==0){
				printf("%d  %d  %d %d \n",Hscan[foundIn], Hscan[foundIn+1], foundIn,medPos);
			      
				int start=max(0,(foundIn-1)<<3);
				int stop=min(256, (foundIn+2)<<3);
				printf("%d %d\n", start,stop);
				for(int i=start; i<stop;i++)
				  printf ("%d ",H[i]);
			      printf("\n"); 			
			      }
*/
		syncthreads();
		if(tx==0){
			int32_t total=Hscan[foundIn];
			int32_t pos=(foundIn+1)<<3;
		        total+=H[pos];     
			
/*			if(blockIdx.x==0 && threadIdx.x==0){
				printf(" %d \n",total );
			} 
*/			*retval=pos+8;
			for(int i=0; i<8;i++)
			{
				if(total>=medPos){
					*retval=pos+i;
					break;
				}
				total+=H[pos+i];
/* 			if(blockIdx.x==0 && threadIdx.x==0){
				printf(" %d \n",total );
			}                            
*/        		}
       		
		}
//	}

     
}
 

/////////////////////////////////////////
/////////////////////////////////////////
/////////////////////////////////////////

__global__ void cuMedianFilter (im_type* src, im_type* dest, hist_type * hist, int32_t rows, int32_t cols, int32_t r, int32_t medPos)
{
    __shared__ hist_type H[MF_HIST_SIZE];
    __shared__ hist_type Hscan[MF_HIST_SIZE];
    __shared__ im_type retval;


    for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
    	hist[j*MF_HIST_SIZE+src[j]]=r+2;
    }
    syncthreads();

    for(int i=1; i< r; i++){
    	int32_t pos=min(i,rows-1);
        for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
        	hist[j*MF_HIST_SIZE+src[pos*cols+j]]++;
        }
        syncthreads();
    }


    for(int i=0; i< rows; i++){
        histogramClear(H); syncthreads();

        int32_t possub=max(0,i-r-1);
        int32_t posadd=min(rows-1,i+r);

        for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
        	hist[MF_HIST_SIZE*j+ src[possub*cols+j] ]--;
        	hist[MF_HIST_SIZE*j+ src[posadd*cols+j] ]++;
        }
        syncthreads();

        for(int32_t j=0;j<(2*r);j++){
        	histogramAdd(H,hist+MF_HIST_SIZE*j); syncthreads();
        }

        for(int32_t j=r;j<cols-r;j++){
            int32_t possub=max(j-r,0);
            int32_t posadd=min(j+r,cols-1);
            histogramAdd(H, hist+posadd*MF_HIST_SIZE); syncthreads();
//            histogramMedian(H,MF_HIST_SIZE,medPos, &retval);
//            histogramMedianPar(H,Hscan,MF_HIST_SIZE,medPos, 8,&retval);
//            histogramMedianPar256(H,Hscan,MF_HIST_SIZE,medPos, 8,&retval);

            histogramMedianPar32WorkInefficient(H,Hscan,MF_HIST_SIZE,medPos, 8,&retval);
//            return;
            syncthreads();

            if(threadIdx.x==0){
            	dest[i*cols+j]=retval;
           }
                syncthreads();

            histogramSub(H, hist+possub*MF_HIST_SIZE); syncthreads();
        }
        syncthreads();
    }
}


__global__ void cuMedianFilterMultiBlock (im_type* src, im_type* dest, hist_type * histPar, int32_t rows, int32_t cols, int32_t r, int32_t medPos)
{
    __shared__ hist_type H[MF_HIST_SIZE];
    __shared__ hist_type Hscan[32];
    __shared__ im_type retval;

    int32_t extraRowThread=rows%gridDim.x;
    int32_t doExtraRow=blockIdx.x<extraRowThread;
    int32_t startRow=0, stopRow=0;
    int32_t rowsPerBlock= rows/gridDim.x+doExtraRow;

    // The following code partitions the work to the blocks. Some blocks will do one row more
	// than other blocks. This code is responsible for doing that balancing
	if(doExtraRow){
        startRow=rowsPerBlock*blockIdx.x;
        stopRow=min(rows, startRow+rowsPerBlock);
    }
    else{
        startRow=(rowsPerBlock+1)*extraRowThread+(rowsPerBlock)*(blockIdx.x-extraRowThread);    
        stopRow=min(rows, startRow+rowsPerBlock);        
    }

    hist_type* hist=histPar+cols*MF_HIST_SIZE*blockIdx.x;
    if (blockIdx.x==(gridDim.x-1))
    	stopRow=rows;
    syncthreads();
    int32_t initNeeded=0, initVal, initStartRow, initStopRow;

    if(blockIdx.x==0){
    	initNeeded=1; initVal=r+2; initStartRow=1;	initStopRow=r;
    }
    else if (startRow<(r+2)){
    	//initNeeded=1; initVal=r+2-startRow-1; initStartRow=1+startRow;	initStopRow=r+startRow+1;
    	initNeeded=1; initVal=r+2-startRow; initStartRow=1;	initStopRow=r+stopRow-startRow;
    }
    else{
    	initNeeded=0; initVal=0; initStartRow=startRow-(r+1);	initStopRow=r+startRow;    	
    }
   syncthreads();
//   int counter=0;
    // In the original algorithm an initialization phase was required as part of the window was outside the
	// image. In this parallel version, the initializtion is required for all thread blocks that part
	// of the median filter is outside the window.
	// For all threads in the block the same code will be executed.
	if (initNeeded){
		for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
			hist[j*MF_HIST_SIZE+src[j]]=initVal;
		}
//		counter+=initVal;
    }
    syncthreads();
    
	// Fot all remaining rows in the median filter, add the values to the the histogram
	for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
		for(int i=initStartRow; i<initStopRow; i++){
			int32_t pos=min(i,rows-1);
				hist[j*MF_HIST_SIZE+src[pos*cols+j]]++;
			}
	}
    
  syncthreads();
//     if(threadIdx.x==0 && initNeeded)
//      printf("%d, %d, %d, %d \n",blockIdx.x, startRow,stopRow, counter);


	 // Going through all the rows that the block is responsible for.
	 int32_t inc=blockDim.x*MF_HIST_SIZE;
     for(int i=startRow; i< stopRow; i++){
         // For every new row that is started the global histogram for the entire window is restarted.
		 histogramClear(H);
		 // Computing some necessary indices
         int32_t possub=max(0,i-r-1),posadd=min(rows-1,i+r);
		 int32_t possubMcols=possub*cols, posaddMcols=posadd*cols;
         syncthreads();
         int32_t histPos=threadIdx.x*MF_HIST_SIZE;
		 // Going through all the elements of a specific row. Foeach histogram, a value is taken out and 
		 // one value is added.
         for (int32_t j=threadIdx.x; j<cols; j+=blockDim.x){
          	hist[histPos+ src[possubMcols+j] ]--;
          	hist[histPos+ src[posaddMcols+j] ]++;
          	histPos+=inc;
			syncthreads();
         }

         
         histogramMultipleAdd(H,hist, 2*r+1);         

         syncthreads();         	
         int32_t rowpos=i*cols;
         int32_t cols_m_1=cols-1;
         for(int32_t j=r;j<cols-r;j++){
             int32_t possub=max(j-r,0);
             int32_t posadd=min(j+1+r,cols_m_1);
             //histogramMedianPar32Multi(H,Hscan,MF_HIST_SIZE,medPos, 8,&retval);
             histogramMedianPar32WorkInefficient(H,Hscan,MF_HIST_SIZE,medPos, 8,&retval);
             syncthreads();
             if (threadIdx.x==0)
            	 dest[rowpos+j]=retval;

             //histogramAddAndSub(H, hist+posadd*MF_HIST_SIZE,hist+possub*MF_HIST_SIZE);                 
             histogramAddAndSub(H, hist+(int)(posadd<<8),hist+(int)(possub<<8));                 
             syncthreads();
        }
         syncthreads();
     }

}


void readImage(char* filename,im_type* imread, int32_t rows, int32_t cols);
float psnr(im_type* im1,im_type* im2, int32_t rows, int32_t cols);


int main(const int argc, char *argv[])
{
	im_type *hostSrc=NULL,*hostDest=NULL, *hostRef;
	im_type *devSrc=NULL,*devDest=NULL;

	int32_t rows=MF_IM_SIZE,cols=MF_IM_SIZE;
	int32_t pixels=rows*cols;
	int32_t memBytesImage=sizeof(im_type)*pixels;

	hist_type* devHist;
	int32_t memBytesHist=sizeof(hist_type)*cols*MF_HIST_SIZE;

	hipSetDevice(1);

	// Allocating host and device memory.
	hostSrc=(im_type*)malloc(memBytesImage);
	hostDest=(im_type*)malloc(memBytesImage);
	hostRef=(im_type*)malloc(memBytesImage);
	CUDA(hipMalloc((void**)(&devSrc), memBytesImage));
	CUDA(hipMalloc((void**)(&devDest), memBytesImage));
	CUDA(hipMalloc((void**)(&devHist), memBytesHist));

	// Loading image from file
	char filename[]="barbara.csv";
	readImage(filename, hostSrc, rows, cols);

	// Copying data from host to device.
	CUDA(hipMemcpy(devSrc,hostSrc,memBytesImage,hipMemcpyHostToDevice));
//	CUDA(hipMemcpy(devDest,hostSrc,memBytesImage,hipMemcpyHostToDevice));
	CUDA(hipMemset(devDest,0,memBytesImage));
	CUDA(hipMemset(devHist,0,memBytesHist));


	int32_t kernel=5;
	int32_t r=(kernel-1)/2;
	int32_t medPos=2*r*r+2*r;

	// Setting CUDA kernel properties.
	dim3 gridDim; gridDim.x=1;
	dim3 blockDim; blockDim.x=256;

	hipEvent_t start,stop; float time;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start,0);
	cuMedianFilter<<<gridDim,blockDim>>>(devSrc, devDest, devHist, rows, cols, r, medPos);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);


	// Copying filtered image back from the device to the host.
	CUDA(hipMemcpy(hostDest,devDest,memBytesImage,hipMemcpyDeviceToHost));

	// Loading reference image for comparison purposes.
	char refFileName[]="ref.csv";
	readImage(refFileName, hostRef, rows, cols);
	float psnrval = psnr(hostDest, hostRef, rows, cols);
	printf("Time: %f  (secs) \t\t PSNR: %f \n",time/1000,psnrval);


	for(int gridSize=32; gridSize<512; gridSize+=32)
	//for(int gridSize=32; gridSize<64; gridSize+=32)
	{
		// Setting CUDA kernel properties.
		dim3 gridDim; gridDim.x=gridSize;
		dim3 blockDim; blockDim.x=64;
		hist_type* devHistMulti;
		int32_t memBytesHistMulti=sizeof(hist_type)*cols*MF_HIST_SIZE*gridDim.x;
		CUDA(hipMalloc((void**)(&devHistMulti), memBytesHistMulti));
		CUDA(hipMemset(devDest,0,memBytesImage));
		CUDA(hipMemset(devHistMulti,0,memBytesHistMulti));

		hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		
		hipEvent_t start,stop; float multiTime;
		hipEventCreate(&start); hipEventCreate(&stop);
		hipEventRecord(start,0);
		cuMedianFilterMultiBlock<<<gridDim,blockDim>>>(devSrc, devDest, devHistMulti, rows, cols, r, medPos);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipDeviceSynchronize();
		hipEventElapsedTime(&multiTime, start, stop);

		hipFree(devHistMulti);

		// Copying filtered image back from the device to the host.
		CUDA(hipMemcpy(hostDest,devDest,memBytesImage,hipMemcpyDeviceToHost));

		float psnrval = psnr(hostDest, hostRef, rows, cols);
		printf("Time: %f  (secs) \t\t PSNR: %f \t\t Speedup %f\n",multiTime/1000,psnrval,time/multiTime);

	}

	// Deallocating host and device memory.
	hipFree(devHist);
	hipFree(devSrc);
	hipFree(devDest);
	free(hostRef);
	free(hostSrc);
	free(hostDest);

    return 0;
}

void readImage(char* filename,im_type* imread, int32_t rows, int32_t cols)
{
	FILE* file=fopen(filename,"r");
	int32_t temp;
	for(int32_t r=0; r<rows;r++){
		for(int32_t c=0; c<cols; c++){
			int read=fscanf(file, "%d ", &temp);
			im_type stam=temp;
			imread[r*cols+c]=(im_type)stam;
		}
	}

	fclose(file);
}

float psnr(im_type* im1,im_type* im2, int32_t rows, int32_t cols){
	int64_t mse=0;
	for(int32_t r=0; r<rows;r++){
		for(int32_t c=0; c<cols; c++){
			int32_t f1=im1[r*cols+c];
			int32_t f2=im2[r*cols+c];
			mse+= (f1-f2)*(f1-f2);
//			printf("%d ",(f1-f2)*(f1-f2));
		}
//		printf("\n");
	}
	if (mse==0)
		return 100;
//	printf("%ld ", mse);
	float fmse=mse/(cols*rows);
	return 20*log10(255/sqrt(fmse) );

}
